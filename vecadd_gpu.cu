#include "hip/hip_runtime.h"
// Based on https://www.olcf.ornl.gov/tutorials/cuda-vector-addition/
#include "mpi.h"
#include "math.h"
#include "stdio.h"
#include "stdlib.h"

extern int DATA_DISTRIBUTE;
extern int DATA_COLLECT;

// CUDA kernel. Each thread takes care of one element of c
__global__ void gpu_vecadd(float *a, float *b, float *c, int n)
{
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;
 
  // Make sure we do not go out of bounds
  if (id < n)
    c[id] = a[id] + b[id];
}

void compute_node_gpu(unsigned int vector_size ) 
{
  int np;
  unsigned int num_bytes = vector_size * sizeof(float);
  float *input_a, *input_b, *output;
  MPI_Status status;
  MPI_Comm_size(MPI_COMM_WORLD, &np);
  int server_process = np - 1;

  // This process id
  int rank = -1;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  // Name the input and output vectors

  // Device (GPU) input vectors
  float *gpu_input_a;
  float *gpu_input_b;
  
  //Device output vector
  float *gpu_output;

  // Allocate memory	

  // Allocate memory for each vector on host
  input_a = (float*)malloc(num_bytes);
  input_b = (float*)malloc(num_bytes);
  output = (float*)malloc(num_bytes);
 
  // Allocate memory for each vector on the GPU
  hipMalloc(&gpu_input_a, num_bytes);
  hipMalloc(&gpu_input_b, num_bytes);
  hipMalloc(&gpu_output, num_bytes);

  /* Get the input data from data server process */
  MPI_Recv(input_a, vector_size, MPI_FLOAT, server_process, DATA_DISTRIBUTE, MPI_COMM_WORLD, &status);

  MPI_Recv(input_b, vector_size, MPI_FLOAT, server_process, DATA_DISTRIBUTE, MPI_COMM_WORLD, &status);

  /* Compute the partial vector addition */

  // Copy host vectors to device
  hipMemcpy( gpu_input_a, input_a, num_bytes, hipMemcpyHostToDevice);
  hipMemcpy( gpu_input_b, input_b, num_bytes, hipMemcpyHostToDevice);
 
  int block_size, grid_size;
 
  // Number of threads in each thread block
  block_size = 1024;
 
  // Number of thread blocks in grid
  grid_size = (int)ceil((float)vector_size/block_size);
 
  // Execute the kernel
  gpu_vecadd<<<grid_size, block_size>>>(gpu_input_a, gpu_input_b, gpu_output, vector_size);

  // Copy array back to host
  hipMemcpy( output, gpu_output, num_bytes, hipMemcpyDeviceToHost );

  // Signal that computation is done
  printf("ComputeNode (%d): GPU partial vector addition complete.\n", rank);
  fflush(stdout);

  // Check GPU calculation against CPU version for debugging
  // Add vectors using CPU
  float *cpu_output = (float *)malloc(num_bytes);
  for(int i = 0; i < vector_size; i++) 
    cpu_output[i] = input_a[i] + input_b[i];

  float error = 0;
  float total = 0;
  for(int i = 0; i < vector_size; i++)
    {
      total = total + cpu_output[i];
      error = error + abs(cpu_output[i] - output[i]);
    }

  printf("ComputeNode (%d): GPU result differs from CPU result by %f%%.\n", rank, 100.0*error/total);
  fflush(stdout);

  printf("ComputeNode (%d): CPU result (first 10 elements):\n", rank);
  for(int i = 0; i < 10; i++)
      printf("%f ", cpu_output[i]);
  printf("\n");
  fflush(stdout);	 

  printf("ComputeNode (%d): GPU result (first 10 elements):\n", rank);
  for(int i = 0; i < 10; i++)
      printf("%f ", output[i]);
  printf("\n");
  fflush(stdout);	 

  MPI_Barrier(MPI_COMM_WORLD);

  // Send the output to the data server
  MPI_Send(output, vector_size, MPI_FLOAT, server_process, DATA_COLLECT, MPI_COMM_WORLD);

  // Clean up memory

  // Release device memory
  hipFree(gpu_input_a);
  hipFree(gpu_input_b);
  hipFree(gpu_output);
 
  // Release host memory
  free(input_a);
  free(input_b);
  free(output);
  free(cpu_output); 
}
